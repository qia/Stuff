#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>

extern "C"

__global__ void rtruncnorm_kernel(float *x, int n, 
                  float *mu, float *sigma, float *a, float *b,
                  int numbtries, int maxtries)
{
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    int idx = myblock * blocksize + subthread;
    hiprandState rng;
    hiprand_init (idx,0,0,&rng);

if ( idx < n) { 
  int accepted = 0 ; // 0 means False , 1 means True
  while ( accepted == 0 && numbtries < maxtries) { 
    numbtries = numbtries + 1;
    x[idx] = mu[idx] + sigma[idx]*hiprand_normal(&rng);
  if (x[idx] >=a[idx] && x[idx]<=b[idx]){
    accepted = 1;
    }
  }

  while ( accepted == 0 ) {
  float u_bar = 0.;
  float psi = 0.;
  if (isinf(b[idx]!=0)) {
  u_bar = a[idx];
  } else {
  u_bar = -b[idx];
  }
  float alpha = (u_bar + sqrt((pow(u_bar,2)+4)))/2;
  float z  = u_bar  - log (hiprand_uniform(&rng)/alpha);
  if (u_bar  < alpha ){
  psi  = exp (-pow(alpha -z ,2)/2);
}else{
  psi = exp(-pow(u_bar -alpha,2)/2)*exp(-pow(alpha-z,2)/2);
}

if (hiprand_uniform(&rng) < psi  ) { 
  if (isinf(b[idx]!=0)) {
  x[idx] = mu[idx] + sigma[idx]*z ;
  accepted = 1;
  }else {
  x[idx] = mu[idx] - sigma[idx]*z ;
  accepted = 1;
  }  
}
}  
}
} // END extern "C"
